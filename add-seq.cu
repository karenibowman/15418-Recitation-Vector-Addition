
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// Normal c++ function. Adds each element pair one at a time.
void vecAdd(double *a, double *b, double *c, int n)
{
    for (int i=0; i<n; i++)
        c[i] = a[i] + b[i];
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;
 
    // input vectors
    double *h_a;
    double *h_b;

    // output vector
    double *h_c;
 
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on the CPU
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    int i;
    // Initialize vectors on CPU 
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Execute addition function
    vecAdd(h_a, h_b, h_c, n);
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);

 
    // Release CPU (host) memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}